#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// called from host, run on device
__global__ void add_gpu(long *in1,long *out)
{
	long idx=threadIdx.x; // flat model
	// out[idx]=in1[idx]+in2[idx];
	out[idx] = in1[2*idx] + in1[2*idx+1];
}
int main()
{
	// pointers to host memory
	long *a,*c;

	// pointers to device memory
	long *a_d,*b_d,*c_d;
	int exp=15;
	long N = (long)pow(2,exp);
	printf("%ld\n",N);
	printf("%d\n",(int)log2((double)N));
	long i;

	// allocate arrays a, b and c on host
	a=(long*)malloc(N*sizeof(long));
	// b=(float*)malloc(N*sizeof(float));
	c=(long*)malloc(N*sizeof(long));

	// allocate arrays a_d, b_d and c_d on device
	hipMalloc((void**)&a_d,sizeof(long)*N);
	// hipMalloc((void**)&b_d,sizeof(float)*N);
	hipMalloc((void**)&c_d,sizeof(long)*N);

	// initialize arrays a and b
	srand(time(NULL));
	for(i=0;i<N;i++){
		a[i]= rand() % 100;
		// b[i]=-(float) i/2.0f;
	}

	// copy input from host memory to device memory
	hipMemcpy(a_d,a,sizeof(long)*N,hipMemcpyHostToDevice);
	// hipMemcpy(b_d,b,sizeof(float)*N,hipMemcpyHostToDevice);

	// execution configuration: How the threads are arranged, FLAT and LINEAR.
	int k;
	/*
	for(i=0;i<N;i++)
		printf("a[%d]=%d\n",i,a[i]);
	printf("\n");
	*/
	for(k = exp-1;k>=0;k--) {
		N /= 2;
		// (long)pow(2,k)
		dim3 dimGrid(1),dimBlock(N);
		add_gpu<<<dimGrid,dimBlock>>>(a_d, c_d);
		// float* tmp = b_d;
		/*
		hipMemcpy(c,c_d,sizeof(int)*N,hipMemcpyDeviceToHost);
		for(i=0;i<(int)pow(2,k);i++)
			printf("c[%d]=%d\n",i,c[i]);
		printf("\n");
		*/
		if(k) {
			b_d = a_d;
			a_d = c_d;
			c_d = b_d;
		}
	}
	
	// copy result from device memory to host memory
	hipMemcpy(c,c_d,sizeof(long)*N,hipMemcpyDeviceToHost);

	printf("%ld\n",c[0]);
	
	free(a);
	// free(b);
	free(c);
	hipFree(a_d);
	// hipFree(b_d);
	hipFree(c_d);
}
