#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// called from host, run on device
__global__ void add_gpu(long *in1,long *out)
{
	long idx=threadIdx.x; // flat model
	out[idx] = in1[2*idx] + in1[2*idx+1];
}
int main()
{
	long *a,*c;
	long *a_d,*b_d,*c_d;
	int exp=15;
	long N = (long)pow(2,exp);
	printf("%ld\n",N);
	printf("%d\n",(int)log2((double)N));
	long i;
	a=(long*)malloc(N*sizeof(long));
	c=(long*)malloc(N*sizeof(long));
	hipMalloc((void**)&a_d,sizeof(long)*N);
	hipMalloc((void**)&c_d,sizeof(long)*N);
	srand(time(NULL));
	for(i=0;i<N;i++){
		a[i]= rand() % 100;
	}
	hipMemcpy(a_d,a,sizeof(long)*N,hipMemcpyHostToDevice);
	int k;
	for(k = exp-1;k>=0;k--) {
		N /= 2;
		dim3 dimGrid(1),dimBlock(N);
		add_gpu<<<dimGrid,dimBlock>>>(a_d, c_d);
		if(k) {
			b_d = a_d;
			a_d = c_d;
			c_d = b_d;
		}
	}
	hipMemcpy(c,c_d,sizeof(long)*N,hipMemcpyDeviceToHost);
	printf("%ld\n",c[0]);
	free(a);
	free(c);
	hipFree(a_d);
	hipFree(c_d);
}
